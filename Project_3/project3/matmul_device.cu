#include "hip/hip_runtime.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include "matmul_device.cuh"

/*
 * Read TODO items below
 */




__global__
void naiveMatmul(float *a, float *b, float *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    float acc = 0;
    for (int k=0; k<n; k++) {
	acc += a[i*n+k] * b[k*n+j];
    }
    c[i*n+j] = acc;
}



__global__ void cacheMatmul(float *a, float *b, float *c, int n)
{

     int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    float acc = 0;
    for(int k1=0;k1<n;k1+=gridDim.x)
    {
    acc=c[i*n+j];
    for(int k=k1;k<k1+gridDim.x;k++)
     {
    	acc += a[i*n+k] * b[k*n+j];
     }
      c[i*n+j] = acc;
     }
}


__global__ void sharedMatmul(float *a, float *b, float *c, int n)
{

    __shared__ float A_tile[32][32];
    __shared__ float B_tile[32][32];
    int width = gridDim.x*blockDim.x;

    float acc = 0;   
    
    int i = blockIdx.x*32 + threadIdx.x;
    int j = blockIdx.y*32 + threadIdx.y;
    

    /* Accumulate C tile by tile. */

    for (int tileIdx = 0; tileIdx < gridDim.x ; tileIdx+=1)
    {

        /* Load one tile of A and one tile of B into shared mem */
    
	A_tile[threadIdx.y][ threadIdx.x] = a[j * width + tileIdx*32+threadIdx.x];  
        B_tile[threadIdx.y][threadIdx.x] = b[(tileIdx * 32 + threadIdx.y)* width+ i ]; 
    
        __syncthreads();                                        

        /* Accumulate one tile of C from tiles of A and B in shared mem */

        for (int k = 0 ;k < 32; k++)
        {   
            acc += A_tile[threadIdx.y][k] * B_tile[k][threadIdx.x];    
        }
    
        __syncthreads();                                                            

    }

    c[j * width + i ] = acc;                            
    
}


void cudaMatmul(float *a, float *b, float *c, int n, MatmulImplementation type)
{
    // TODO: play with the gridSize and blockSize to find the best one
    if (type == NAIVE) {
        dim3 blockSize(32, 32);
        dim3 gridSize(n / 32, n / 32);
        naiveMatmul<<<gridSize, blockSize>>>(a,b,c,n);
    }
    else if (type == CACHE) {
        dim3 blockSize(32, 32);
        dim3 gridSize(n / 32, n / 32);
        cacheMatmul<<<gridSize, blockSize>>>(a,b,c,n);
    }
    else if (type == SHARED) {
        dim3 blockSize(32, 32);
        dim3 gridSize(n / 32, n / 32);
        sharedMatmul<<<gridSize, blockSize>>>(a,b,c,n);
    }
    // Unknown type
    else
        assert(false);
}
